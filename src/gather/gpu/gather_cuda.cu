#include "hip/hip_runtime.h"
#include <cuda/std/span>
#include <hip/hip_fp16.h>
#include <iostream>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void gather_kernel(
  cuda::std::span<const T> data,
  cuda::std::span<const int> indices,
  cuda::std::span<T> output,
  int axis,
  int data_shape_axis,
  int output_size
) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < output_size) {
    int data_idx = 0;
    if (axis == 0) {
      int row = indices[idx / data_shape_axis];
      int col = idx % data_shape_axis;
      data_idx = row * data_shape_axis + col;
    } else if (axis == 1) {
      int row = idx / indices.size();        
      int col = indices[idx % indices.size()]; 
      data_idx = row * data_shape_axis + col; 
    }
    output[idx] = data[data_idx];
  }
}

extern "C" {
  void gather_cuda_f16(
    void const *data, void const *indices, void *output,
    int axis, int data_shape_axis, int output_size, int indices_size) {
    int blockSize = 256;
    int gridSize = (output_size + blockSize - 1) / blockSize;
    gather_kernel<<<gridSize, blockSize>>>(
      cuda::std::span<const __half>((const __half *)data, data_shape_axis * 3),
      cuda::std::span<const int>((const int *)indices, indices_size),
      cuda::std::span<__half>((__half *)output, output_size),
      axis,
      data_shape_axis,
      output_size
    );
  }


  void gather_cuda_f32(
    void const *data, void const *indices, void *output,
    int axis, int data_shape_axis, int output_size, int indices_size) {
    int blockSize = 256;
    int gridSize = (output_size + blockSize - 1) / blockSize;
    gather_kernel<<<gridSize, blockSize>>>(
      cuda::std::span<const float>((const float *)data, data_shape_axis * 3),
      cuda::std::span<const int>((const int *)indices, indices_size),
      cuda::std::span<float>((float *)output, output_size),
      axis,
      data_shape_axis,
      output_size
    );
  }
}

